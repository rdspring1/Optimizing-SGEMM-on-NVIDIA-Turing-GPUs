#include "utils.cuh"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <helper_string.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#define MYSGEMM mysgemm_naive // select the kernel here

int main(int argc, char **argv) {
  if (argc != 2) {
    printf("Please select a kernel (range 0 - 11).\t"
           "0 is for NVIDIA cuBLAS).\n");
    exit(-1);
  }

  int SIZE[24];
  for (int i = 0; i < 24; i++) {
    SIZE[i] = (i + 1) << 8;
  }

  int kernel_num = atoi(argv[1]);
  if (kernel_num < 0 || kernel_num > 11) {
    printf("Please enter a valid kernel number (0-11).\n");
    exit(-2);
  }
  int m, n, k, max_size;
  int n_count, N = 10, upper_limit;
  if (kernel_num <= 4 && kernel_num != 0) {
    upper_limit = 8;
  } else {
    upper_limit = (sizeof(SIZE) / sizeof(int));
  }
  max_size = SIZE[upper_limit - 1];
  float *A = NULL, *B = NULL, *C = NULL, *C_ref = NULL;     // host matrices
  float *dA = NULL, *dB = NULL, *dC = NULL, *dC_ref = NULL; // device matrices
  float alpha = 1.0, beta = 0.; // two arbitary input parameters

  float elapsed_time;
  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);

  hipEvent_t beg, end;
  hipEventCreate(&beg);
  hipEventCreate(&end);

  A = (float *)malloc(sizeof(float) * max_size * max_size);
  B = (float *)malloc(sizeof(float) * max_size * max_size);
  C = (float *)malloc(sizeof(float) * max_size * max_size);
  C_ref = (float *)malloc(sizeof(float) * max_size * max_size);

  randomize_matrix(A, max_size * max_size);
  randomize_matrix(B, max_size * max_size);
  randomize_matrix(C, max_size * max_size);

  copy_matrix(C, C_ref, max_size * max_size);

  CUDA_CALLER(hipMalloc((void **)&dA, sizeof(float) * max_size * max_size));
  CUDA_CALLER(hipMalloc((void **)&dB, sizeof(float) * max_size * max_size));
  CUDA_CALLER(hipMalloc((void **)&dC, sizeof(float) * max_size * max_size));
  CUDA_CALLER(
      hipMalloc((void **)&dC_ref, sizeof(float) * max_size * max_size));
  CUDA_CALLER(hipMemcpy(dA, A, sizeof(float) * max_size * max_size,
                         hipMemcpyHostToDevice));
  CUDA_CALLER(hipMemcpy(dB, B, sizeof(float) * max_size * max_size,
                         hipMemcpyHostToDevice));
  CUDA_CALLER(hipMemcpy(dC, C, sizeof(float) * max_size * max_size,
                         hipMemcpyHostToDevice));
  CUDA_CALLER(hipMemcpy(dC_ref, C_ref, sizeof(float) * max_size * max_size,
                         hipMemcpyHostToDevice));

  for (int i_count = 0; i_count < upper_limit; i_count++) {
    m = n = k = SIZE[i_count];
    printf("\nM=N=K=%d:\n", m);
    if (kernel_num != 0) { // not cuBLAS
      hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, dA,
                  m, dB, k, &beta, dC_ref, m);
      test_kernel(kernel_num, m, n, k, alpha, dA, dB, beta, dC);
      hipDeviceSynchronize();
      hipMemcpy(C, dC, sizeof(float) * m * n, hipMemcpyDeviceToHost);
      hipMemcpy(C_ref, dC_ref, sizeof(float) * m * n, hipMemcpyDeviceToHost);
      hipDeviceSynchronize();
      if (!verify_matrix(C_ref, C, m * n)) {
        printf("Failed to pass the correctness verification against NVIDIA "
               "cuBLAS. Exited.\n");
        exit(-3);
      }
    }

    hipEventRecord(beg);
    if (kernel_num != 0) {
      for (n_count = 0; n_count < N; n_count++) {
        test_kernel(kernel_num, m, n, k, alpha, dA, dB, beta, dC);
      }
    } else {
      for (n_count = 0; n_count < N; n_count++) {
        test_kernel(kernel_num, m, n, k, alpha, dA, dB, beta, dC,
                    cublas_handle);
      }
    }

    hipEventRecord(end);
    hipEventSynchronize(beg);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, beg, end);
    elapsed_time /= 1000.;

    printf("Average elasped time: %f second, performance: %f GFLOPS.\n",
           elapsed_time / N, 2. * 1e-9 * N * m * n * k / elapsed_time);
    fflush(stdout);
    copy_matrix(C_ref, C,
                m * n); // sync C with cuBLAS to prepare for the next run
  }
  hipDeviceSynchronize();

  free(A);
  free(B);
  free(C);
  free(C_ref);

  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
  hipFree(dC_ref);
  hipDeviceSynchronize();
  return 0;
}
